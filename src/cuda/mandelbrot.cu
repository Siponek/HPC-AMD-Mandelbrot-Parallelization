#include "hip/hip_runtime.h"
#include <LogUtils.h>
#include <chrono>
#include <complex>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <string>

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

// Alias to the global namespace for CUDA functions
namespace cuda
{
hipError_t malloc(void **devPtr, size_t size)
{
	return hipMalloc(devPtr, size);
}

hipError_t free(void *devPtr) { return hipFree(devPtr); }
} // namespace cuda

namespace MandelbrotSet
{
// Ranges of the set
constexpr float MIN_X = -2.0;
constexpr float MAX_X = 1.0;
constexpr float MIN_Y = -1.0;
constexpr float MAX_Y = 1.0;

// Image ratio
constexpr float RATIO_X = (MAX_X - MIN_X);
constexpr float RATIO_Y = (MAX_Y - MIN_Y);

} // namespace MandelbrotSet
namespace fs = std::filesystem;

using namespace std;
using namespace MandelbrotSet;

__host__ void check_cuda_errors(hipError_t err_sync,
								hipError_t err_async)
{
	if (err_sync != hipSuccess)
	{
		cerr << "Sync kernel error: "
			 << hipGetErrorString(err_sync) << endl;
		exit(-1);
	}
	if (err_async != hipSuccess)
	{
		cerr << "Async kernel error: "
			 << hipGetErrorString(err_async) << endl;
		exit(-1);
	}
}

__host__ void log_execution_details(
	const string &log_file, const string &file_name, int iterations,
	int resolution_value, int WIDTH, int HEIGHT, double STEP,
	int cuda_threads_used, chrono::duration<double> duration)
{
	ofstream log(log_file, ios::app);
	if (log.is_open())
	{
		log << "Date:\t" << __DATE__ << " " << __TIME__
			<< "\tProgram:\t" << file_name << "\t\tIterations:\t"
			<< iterations << "\tResolution:\t" << resolution_value
			<< "\tWidth:\t" << WIDTH << "\tHeight:\t" << HEIGHT
			<< "\tStep:\t" << STEP << "\tCUDA threads:\t"
			<< cuda_threads_used << "\tTime:\t" << duration.count()
			<< "\tseconds" << endl;
		log.close();
	}
	else
	{
		cerr << "Unable to open log file." << endl;
	}
}

__device__ int dev_Mandelbrot_kernel(int collumn, int row,
									 double step, int minX,
									 int minY, int iterations)
{
	int count = 0;

	hipDoubleComplex c = make_hipDoubleComplex(minX + collumn * step,
											 minY + row * step);
	hipDoubleComplex z = make_hipDoubleComplex(0, 0);
	while (hipCabs(z) < 2.0 && count < iterations)
	{
		z = hipCadd(hipCmul(z, z), c);
		count++;
	}

	return (count < iterations) ? count : 0;
}

__global__ void mandelbrotKernel(int *image, double step, int minX,
								 int minY, int iterations,
								 int WIDTH, int HEIGHT)
{
	// idiomatic CUDA loop
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if (col >= WIDTH || row >= HEIGHT)
		return;

	int index = row * WIDTH + col;
	image[index] = dev_Mandelbrot_kernel(col, row, step, minX, minY,
										 iterations);
}

int main(int argc, char **argv)
{
	fs::path file_path = argv[0];
	string file_name = file_path.filename().string();
	if (argc < 3)
	{
		cout << "Usage: " << file_name
			 << " <output_file> --iterations <iterations> "
				"--resolution <resolution> --threads <threads>"
			 << endl;
		return -1;
	}

	// Parse command line arguments
	cmdParse::ParsedArgs args =
		cmdParse::parse_cmd_arguments(argc, argv);
	const int iterations = args.iterations;
	const int resolution_value = args.resolution;
	fs::path output_file_path(args.output_file);
	const int cuda_threads_used = args.threads_num;

	// Image size
	const int WIDTH = static_cast<int>(RATIO_X * resolution_value);
	const int HEIGHT = static_cast<int>(RATIO_Y * resolution_value);

	const float STEP = RATIO_X / WIDTH;
	const size_t image_size = HEIGHT * WIDTH;
	unique_ptr<int[]> image(new int[image_size]);

	fill_n(image.get(), image_size, -1);
	int *device_image;
	size_t free_mem, total_mem;
	hipMemGetInfo(&free_mem, &total_mem);

	size_t required_mem = HEIGHT * WIDTH * sizeof(int);
	if (required_mem > free_mem)
	{
		cerr << "Error: Not enough memory on the device." << endl;
		return -1;
	}
	if (cuda::malloc((void **)&device_image, required_mem) !=
		hipSuccess)
	{
		cerr << "Error allocating memory on device." << endl;
		return -33;
	}

	hipMemcpy(device_image, image.get(), image_size * sizeof(int),
			   hipMemcpyHostToDevice);
	dim3 threads_per_block(cuda_threads_used, cuda_threads_used);
	dim3 blocks_per_grid(
		(WIDTH + threads_per_block.x - 1) / threads_per_block.x,
		(HEIGHT + threads_per_block.y - 1) / threads_per_block.y);

	cout << "Image size: " << WIDTH << "x" << HEIGHT << endl;
	cout << "Calculating Mandelbrot set with " << cuda_threads_used
		 << " threads with " << iterations << " iterations." << endl
		 << "blocksize: " << blocks_per_grid.x << " "
		 << blocks_per_grid.y
		 << " threads_per_block: " << threads_per_block.x << " "
		 << threads_per_block.y << endl;

	const auto start = std::chrono::steady_clock::now();
	mandelbrotKernel<<<blocks_per_grid, threads_per_block>>>(
		device_image, STEP, MIN_X, MIN_Y, iterations, WIDTH,
		HEIGHT);
	hipError_t err_sync = hipGetLastError();
	hipError_t err_async = hipDeviceSynchronize();
	check_cuda_errors(hipGetLastError(), hipDeviceSynchronize());
	hipMemcpy(image.get(), device_image, image_size * sizeof(int),
			   hipMemcpyDeviceToHost);

	const auto end = std::chrono::steady_clock::now();
	cuda::free(device_image);
	if (any_of(image.get(), image.get() + image_size,
			   [](int val) { return val == -1; }))
	{
		cerr << "Error: Not all pixels were calculated." << endl;
		return -3;
	}

	chrono::duration<double> duration = end - start;
	cout << endl
		 << "Time elapsed: " << duration.count() << " seconds."
		 << endl;

	//? CSV
	const string additinonalName = "_cuda_";
	const string csvFile = logutils::createCsvFilename(
		output_file_path, additinonalName);
	const string header =
		"DateTime,Program,Iterations,Resolution,"
		"Width,Height,Step,CUDAThreads,Time (seconds)";
	bool has_header = logutils::csvFileHasHeader(csvFile, header);
	ofstream csv(csvFile, ios::app);
	if (csv.is_open())
	{
		if (!has_header)
		{
			cout << "Adding header to csv file." << endl;
			csv << header << endl;
		}
		csv << logutils::getCurrentTimestamp() << "," << file_name
			<< "," << iterations << "," << resolution_value << ","
			<< WIDTH << "," << HEIGHT << "," << STEP << ","
			<< cuda_threads_used << "," << duration.count() << endl;
		csv.close();
		cout << "CSV entry added successfully." << endl;
	}
	else
	{
		cerr << "Unable to open CSV file." << endl;
	}
	//? Log
	const string log_file =
		logutils::create_log_file_name(output_file_path);
	ofstream log(log_file, ios::app);
	log_execution_details(log_file, file_name, iterations,
						  resolution_value, WIDTH, HEIGHT, STEP,
						  cuda_threads_used, duration);
	//
	//? Logging the execution details
	//
	try
	{
		fs::create_directories(output_file_path.parent_path());
	}
	catch (const fs::filesystem_error &e)
	{
		cout << "Error creating directories: " << e.what() << endl;
		return -13;
	}
	// Write the result to a file
	string new_name = to_string(cuda_threads_used) + "_threads_" +
					  to_string(iterations) + "_iterations_" +
					  to_string(resolution_value) + "_resolution";
	// Get the original filename stem and extension
	string filename_stem = output_file_path.stem().string();
	string extension = output_file_path.extension().string();
	string new_filename =
		filename_stem + "_" + new_name + extension;
	// Update the output_file_path with the new filename
	output_file_path =
		output_file_path.parent_path() / new_filename;
	ofstream matrix_out(output_file_path, ios::trunc);
	cout << "Writing to file: " << output_file_path << endl;
	if (!matrix_out.is_open())
	{
		cout << "Unable to open file." << endl;
		return -14;
	}

	for (int row = 0; row < HEIGHT; row++)
	{
		for (int col = 0; col < WIDTH; col++)
		{
			matrix_out << image[row * WIDTH + col];

			if (col < WIDTH - 1)
				matrix_out << ',';
		}
		if (row < HEIGHT - 1)
			matrix_out << endl;
	}
	matrix_out.close();
	image.reset(); // It's here for coding style, but useless
	// delete[] image; // It's here for coding style, but useless
	return 0;
}