#include "hip/hip_runtime.h"
#include <omp.h>

#include <LogUtils.h>
#include <chrono>
#include <complex>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <string>
#include <complex>

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

// Alias to the global namespace for CUDA functions
namespace cuda
{
hipError_t malloc(void **devPtr, size_t size)
{
	return hipMalloc(devPtr, size);
}

hipError_t free(void *devPtr) { return hipFree(devPtr); }
} // namespace cuda

namespace MandelbrotSet
{
// Ranges of the set
constexpr float MIN_X = -2.0;
constexpr float MAX_X = 1.0;
constexpr float MIN_Y = -1.0;
constexpr float MAX_Y = 1.0;

// Image ratio
constexpr float RATIO_X = (MAX_X - MIN_X);
constexpr float RATIO_Y = (MAX_Y - MIN_Y);

#ifndef RESOLUTION
#define RESOLUTION 1000
#endif
constexpr int RESOLUTION_VALUE = (int)RESOLUTION;
// Image size
constexpr int WIDTH = static_cast<int>(RATIO_X * RESOLUTION_VALUE);
constexpr int HEIGHT = static_cast<int>(RATIO_Y * RESOLUTION_VALUE);

constexpr float STEP = RATIO_X / WIDTH;
} // namespace MandelbrotSet
namespace fs = std::filesystem;

#define SCHEDULING_STRING "RUNTIME"
#define SCHEDULING_TYPE runtime
#ifdef DYNAMIC_SCHED
#undef SCHEDULING_STRING
#undef SCHEDULING_TYPE
#define SCHEDULING_TYPE dynamic
#define SCHEDULING_STRING "DYNAMIC"
#elif defined(STATIC_SCHED)
#undef SCHEDULING_STRING
#undef SCHEDULING_TYPE
#define SCHEDULING_TYPE static
#define SCHEDULING_STRING "STATIC"
#elif defined(GUIDED_SCHED)
#undef SCHEDULING_STRING
#undef SCHEDULING_TYPE
#define SCHEDULING_TYPE guided
#define SCHEDULING_STRING "GUIDED"
#endif

using namespace std;
using namespace MandelbrotSet;

int getThreadsUsed(int argc, char **argv)
{
	int threads_used = omp_get_max_threads();
	for (int i = 1; i < argc; ++i)
	{
		string arg = argv[i];
		if (arg == "-threads" && i + 1 < argc)
		{
			threads_used = stoi(argv[++i]);
			if (threads_used <= 0)
			{
				cerr << "Please specify a positive number of "
						"threads."
					 << endl;
				exit(-4);
			}
			omp_set_num_threads(threads_used);
		}
	}
	return threads_used;
}

__device__ int devMandelbrot(int col, int row, double step,
							 int minX, int minY, int iterations)
{
	// hipDoubleComplex c =
	// 	make_hipDoubleComplex(minX + col * step, minY + row * step);
	// hipDoubleComplex z = make_hipDoubleComplex(0, 0);
	const complex<double> c(minX + col * step, minY + row * step);
	complex<double> z(0, 0);
	int count = 0;
	while (abs(z) < 2.0 && count < iterations)
	{
		// z = hipCadd(hipCmul(z, z),
		// 		   c);
		z = pow(z, 2) + c;
		count++;
	}

	return (count < iterations) ? count : 0;
}

__global__ void mandelbrotKernel(int *image, double step, int minX,
								 int minY, int iterations,
								 int WIDTH, int HEIGHT)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (col >= WIDTH || row >= HEIGHT)
		return;

	int index = row * WIDTH + col;

	image[index] =
		devMandelbrot(col, row, step, minX, minY, iterations);
}

int main(int argc, char **argv)
{
	// cout.sync_with_stdio(false);
	fs::path filePath = argv[0];
	string fileName = filePath.filename().string();
	if (argc < 3)
	{
		cout << "Usage: " << fileName
			 << " <output_file> <iterations>" << endl;
		return -1;
	}
	if (argc < 2)
	{
		cout << "Please specify the output file as a parameter."
			 << endl;
		return -1;
	}
	const int iterations = stoi(argv[2]);
	if (iterations <= 0)
	{
		cout << "Please specify a positive number of iterations."
			 << endl;
		return -2;
	}
	const int threads_used = getThreadsUsed(argc, argv);
	int *const image = new int[HEIGHT * WIDTH];
	const size_t image_size = HEIGHT * WIDTH;
	// const size_t image_size = HEIGHT * WIDTH * sizeof(int);
	fill_n(image, image_size, -1);
	int *device_image;
	// cuda::malloc((void **)&device_image, HEIGHT * WIDTH *
	// sizeof(int));
	hipError_t	err = cuda::malloc((void **)&device_image, HEIGHT * WIDTH * sizeof(int));
	
	if (err != hipSuccess || device_image == nullptr)
	{
		cerr << "Error allocating memory on device." << endl;
		return -33;
	}
	hipMemcpy(device_image, image, image_size,
			   hipMemcpyHostToDevice);
	dim3 blockSize(threads_used, threads_used);
	dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x,
				  (HEIGHT + blockSize.y - 1) / blockSize.y);

	cout << "Calculating Mandelbrot set with " << threads_used
		 << " threads with " << iterations << " iterations."
		 << endl;

	const auto start = std::chrono::steady_clock::now();
	mandelbrotKernel<<<gridSize, blockSize>>>(
		device_image, STEP, MIN_X, MIN_Y, iterations, WIDTH,
		HEIGHT);
	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess) {
		fprintf(stderr, "Sync kernel error: %s\n", hipGetErrorString(errSync));
		return -1;
	}
	if (errAsync != hipSuccess) {
		fprintf(stderr, "Async kernel error: %s\n", hipGetErrorString(errAsync));
		return -1;
	}
	hipMemcpy(image, device_image, image_size,
			   hipMemcpyDeviceToHost);
	const auto end = std::chrono::steady_clock::now();

	chrono::duration<double> duration = end - start;
	cout << endl
		 << "Time elapsed: " << duration.count() << " seconds."
		 << endl;

	const string logFile =
		logutils::createLogFileName(argv[1], "_openmp_");
	ofstream log(logFile, ios::app);
	if (log.is_open())
	{
		log << "Date:\t" << __DATE__ << " " << __TIME__
			<< "\tProgram:\t" << fileName << "\t\tIterations:\t"
			<< iterations << "\tResolution:\t" << RESOLUTION_VALUE
			<< "\tWidth:\t" << WIDTH << "\tHeight:\t" << HEIGHT
			<< "\tStep:\t" << STEP << "\tScheduling:\t"
			<< SCHEDULING_STRING << "\tThreads:\t" << threads_used
			<< "\tTime:\t" << duration.count() << "\tseconds"
			<< endl;
		log.close();
	}
	else
	{
		cerr << "Unable to open log file." << endl;
	}

	const fs::path outputFilePath(argv[1]);
	try
	{
		fs::create_directories(outputFilePath.parent_path());
	}
	catch (const fs::filesystem_error &e)
	{
		cout << "Error creating directories: " << e.what() << endl;
		return -13;
	}
	// Write the result to a file
	ofstream matrix_out(outputFilePath, ios::trunc);
	cout << "Writing to file: " << argv[1] << endl;
	if (!matrix_out.is_open())
	{
		cout << "Unable to open file." << endl;
		return -14;
	}

	for (int row = 0; row < HEIGHT; row++)
	{
		for (int col = 0; col < WIDTH; col++)
		{
			matrix_out << image[row * WIDTH + col];

			if (col < WIDTH - 1)
				matrix_out << ',';
		}
		if (row < HEIGHT - 1)
			matrix_out << endl;
	}
	matrix_out.close();

	delete[] image; // It's here for coding style, but useless
	return 0;
}