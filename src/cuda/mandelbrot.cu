#include "hip/hip_runtime.h"
#include <omp.h>

#include <LogUtils.h>
#include <chrono>
#include <complex>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <string>

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

// Alias to the global namespace for CUDA functions
namespace cuda
{
hipError_t malloc(void **devPtr, size_t size)
{
	return hipMalloc(devPtr, size);
}

hipError_t free(void *devPtr) { return hipFree(devPtr); }
} // namespace cuda

namespace MandelbrotSet
{
// Ranges of the set
constexpr float MIN_X = -2.0;
constexpr float MAX_X = 1.0;
constexpr float MIN_Y = -1.0;
constexpr float MAX_Y = 1.0;

// Image ratio
constexpr float RATIO_X = (MAX_X - MIN_X);
constexpr float RATIO_Y = (MAX_Y - MIN_Y);

#ifndef RESOLUTION
#define RESOLUTION 1000
#endif
constexpr int RESOLUTION_VALUE = (int)RESOLUTION;
// Image size
constexpr int WIDTH = static_cast<int>(RATIO_X * RESOLUTION_VALUE);
constexpr int HEIGHT = static_cast<int>(RATIO_Y * RESOLUTION_VALUE);

constexpr float STEP = RATIO_X / WIDTH;
} // namespace MandelbrotSet
namespace fs = std::filesystem;

#define SCHEDULING_STRING "RUNTIME"
#define SCHEDULING_TYPE runtime
#ifdef DYNAMIC_SCHED
#undef SCHEDULING_STRING
#undef SCHEDULING_TYPE
#define SCHEDULING_TYPE dynamic
#define SCHEDULING_STRING "DYNAMIC"
#elif defined(STATIC_SCHED)
#undef SCHEDULING_STRING
#undef SCHEDULING_TYPE
#define SCHEDULING_TYPE static
#define SCHEDULING_STRING "STATIC"
#elif defined(GUIDED_SCHED)
#undef SCHEDULING_STRING
#undef SCHEDULING_TYPE
#define SCHEDULING_TYPE guided
#define SCHEDULING_STRING "GUIDED"
#endif

using namespace std;
using namespace MandelbrotSet;

__host__ int get_threads_used(int argc, char **argv)
{
	int _threads_used = omp_get_max_threads();
	for (int i = 1; i < argc; ++i)
	{
		string arg = argv[i];
		if (arg == "-threads" && i + 1 < argc)
		{
			_threads_used = stoi(argv[++i]);
			if (_threads_used <= 0)
			{
				cerr << "Please specify a positive number of "
						"threads."
					 << endl;
				exit(-4);
			}
			omp_set_num_threads(_threads_used);
		}
	}
	return _threads_used;
}

__host__ void check_cuda_errors(hipError_t err_sync,
								hipError_t err_async)
{
	if (err_sync != hipSuccess)
	{
		cerr << "Sync kernel error: "
			 << hipGetErrorString(err_sync) << endl;
		exit(-1);
	}
	if (err_async != hipSuccess)
	{
		cerr << "Async kernel error: "
			 << hipGetErrorString(err_async) << endl;
		exit(-1);
	}
}

__host__ void log_execution_details(
	const string &logFile, const string &file_name, int iterations,
	int RESOLUTION_VALUE, int WIDTH, int HEIGHT, double STEP,
	int _threads_used, chrono::duration<double> duration)
{
	ofstream log(logFile, ios::app);
	if (log.is_open())
	{
		log << "Date:\t" << __DATE__ << " " << __TIME__
			<< "\tProgram:\t" << file_name << "\t\tIterations:\t"
			<< iterations << "\tResolution:\t" << RESOLUTION_VALUE
			<< "\tWidth:\t" << WIDTH << "\tHeight:\t" << HEIGHT
			<< "\tStep:\t" << STEP << "\tScheduling:\t"
			<< SCHEDULING_STRING << "\tThreads:\t" << _threads_used
			<< "\tTime:\t" << duration.count() << "\tseconds"
			<< endl;
		log.close();
	}
	else
	{
		cerr << "Unable to open log file." << endl;
	}
}

__device__ int dev_Mandelbrot_kernel(int collumn, int row,
									 double step, int minX,
									 int minY, int iterations)
{
	int count = 0;

	hipDoubleComplex c = make_hipDoubleComplex(minX + collumn * step,
											 minY + row * step);
	hipDoubleComplex z = make_hipDoubleComplex(0, 0);
	while (hipCabs(z) < 2.0 && count < iterations)
	{
		z = hipCadd(hipCmul(z, z), c);
		count++;
	}
	// const complex<double> c(minX + collumn * step, minY + row *
	// step); complex<double> z(0, 0); while (abs(z) < 2.0 && count
	// < iterations)
	// {
	// 	z = pow(z, 2) + c;
	// 	count++;
	// }

	return (count < iterations) ? count : 0;
}

__global__ void mandelbrotKernel(int *image, double step, int minX,
								 int minY, int iterations,
								 int WIDTH, int HEIGHT)
{
	// idiomatic CUDA loop
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if (col >= WIDTH || row >= HEIGHT)
		return;

	int index = row * WIDTH + col;
	image[index] = dev_Mandelbrot_kernel(col, row, step, minX, minY,
										 iterations);
}

int main(int argc, char **argv)
{
	// cout.sync_with_stdio(false);
	fs::path file_path = argv[0];
	string file_name = file_path.filename().string();
	if (argc < 3)
	{
		cout << "Usage: " << file_name
			 << " <output_file> <iterations>" << endl;
		return -1;
	}
	if (argc < 2)
	{
		cout << "Please specify the output file as a parameter."
			 << endl;
		return -1;
	}
	const int iterations = stoi(argv[2]);
	if (iterations <= 0)
	{
		cout << "Please specify a positive number of iterations."
			 << endl;
		return -2;
	}
	const int _threads_used = get_threads_used(argc, argv);
	const size_t image_size = HEIGHT * WIDTH;
	unique_ptr<int[]> image(new int[image_size]);

	fill_n(image.get(), image_size, -1);
	int *device_image;
	size_t free_mem, total_mem;
	hipMemGetInfo(&free_mem, &total_mem);

	size_t required_mem = HEIGHT * WIDTH * sizeof(int);
	if (required_mem > free_mem)
	{
		cerr << "Error: Not enough memory on the device." << endl;
		return -1;
	}
	if (cuda::malloc((void **)&device_image, required_mem) !=
		hipSuccess)
	{
		cerr << "Error allocating memory on device." << endl;
		return -33;
	}

	hipMemcpy(device_image, image.get(), image_size * sizeof(int),
			   hipMemcpyHostToDevice);
	dim3 threads_per_block(16, 16);
	dim3 blocks_per_grid(
		(WIDTH + threads_per_block.x - 1) / threads_per_block.x,
		(HEIGHT + threads_per_block.y - 1) / threads_per_block.y);

	cout << "Image size: " << WIDTH << "x" << HEIGHT << endl;
	cout << "Calculating Mandelbrot set with " << _threads_used
		 << " threads with " << iterations << " iterations." << endl
		 << "blocksize: " << blocks_per_grid.x << " "
		 << blocks_per_grid.y
		 << " threads_per_block: " << threads_per_block.x << " "
		 << threads_per_block.y << endl;

	const auto start = std::chrono::steady_clock::now();
	mandelbrotKernel<<<blocks_per_grid, threads_per_block>>>(
		device_image, STEP, MIN_X, MIN_Y, iterations, WIDTH,
		HEIGHT);
	hipError_t err_sync = hipGetLastError();
	hipError_t err_async = hipDeviceSynchronize();
	check_cuda_errors(hipGetLastError(), hipDeviceSynchronize());
	hipMemcpy(image.get(), device_image, image_size * sizeof(int),
			   hipMemcpyDeviceToHost);

	const auto end = std::chrono::steady_clock::now();
	cuda::free(device_image);
	if (any_of(image.get(), image.get() + image_size,
			   [](int val) { return val == -1; }))
	{
		cerr << "Error: Not all pixels were calculated." << endl;
		return -3;
	}

	chrono::duration<double> duration = end - start;
	cout << endl
		 << "Time elapsed: " << duration.count() << " seconds."
		 << endl;

	const string log_file =
		logutils::createLogFileName(argv[1], "_openmp_");
	ofstream log(log_file, ios::app);
	log_execution_details(log_file, file_name, iterations,
						  RESOLUTION_VALUE, WIDTH, HEIGHT, STEP,
						  _threads_used, duration);
	//
	//? Logging the execution details
	//
	const fs::path outputFilePath(argv[1]);
	try
	{
		fs::create_directories(outputFilePath.parent_path());
	}
	catch (const fs::filesystem_error &e)
	{
		cout << "Error creating directories: " << e.what() << endl;
		return -13;
	}
	// Write the result to a file
	ofstream matrix_out(outputFilePath, ios::trunc);
	cout << "Writing to file: " << argv[1] << endl;
	if (!matrix_out.is_open())
	{
		cout << "Unable to open file." << endl;
		return -14;
	}

	for (int row = 0; row < HEIGHT; row++)
	{
		for (int col = 0; col < WIDTH; col++)
		{
			matrix_out << image[row * WIDTH + col];

			if (col < WIDTH - 1)
				matrix_out << ',';
		}
		if (row < HEIGHT - 1)
			matrix_out << endl;
	}
	matrix_out.close();
	image.reset(); // It's here for coding style, but useless
	// delete[] image; // It's here for coding style, but useless
	return 0;
}